#include "hip/hip_runtime.h"
#include <stdio.h> 
#include <stdlib.h> 
#include <hip/hip_runtime.h>
#include <vector>
#include <cstdlib>

#define BLOCK_DIM 512
#define COARSE_FACTOR 1

using namespace std;

//assuming size of array is twice the size of BLOCK_DIM;

__global__ void reduce_kernel(float* input, float* partialSums, unsigned int N){
	unsigned int segment = blockIdx.x * blockDim.x * 2;
	unsigned int i = segment + threadIdx.x * 2;
	for (unsigned int stride = 1; stride <= BLOCK_DIM; stride*=2){
		if(threadIdx.x%stride==0){
			input[i] += input[i+stride];
		}
		__syncthreads();
	} 
	if(threadIdx.x == 0){
		partialSums[blockIdx.x] = input[i];
	}
}

__global__ void reduce_kernel_controldiv(float* input, float* partialSums, unsigned int N){
	unsigned int segment =blockIdx.x * blockDim.x *2;
	unsigned int i = segment + threadIdx.x;
	for(unsigned int stride = BLOCK_DIM; stride >0;stride /=2){
		if(threadIdx.x < stride)
		{
			input[i] +=input[i+stride];
		}
		__syncthreads();
	}
	if(threadIdx.x==0){
		partialSums[blockIdx.x] = input[i];
	}
}

__global__ void reduce_kernel_sharedmem(float* input, float* partialSums, unsigned int N){
	unsigned int segment =blockIdx.x * blockDim.x *2;
	unsigned int i = segment + threadIdx.x;
	__shared__ float input_s[BLOCK_DIM];
	input_s[threadIdx.x] = input[i] + input[i+BLOCK_DIM];
	__syncthreads();
	for(unsigned int stride = BLOCK_DIM/2; stride >0; stride /=2){
		if(threadIdx.x < stride)
		{
			input_s[threadIdx.x] +=input_s[threadIdx.x+stride];
		}
		__syncthreads();
	}
	if(threadIdx.x==0){
		partialSums[blockIdx.x] = input_s[threadIdx.x];
	}
}

__global__ void reduce_kernel_sharedmem_coarsefactor(float* input, float* partialSums, unsigned int N){
	unsigned int segment =blockIdx.x * blockDim.x *2 * COARSE_FACTOR;
	unsigned int i = segment + threadIdx.x;
	__shared__ float input_s[BLOCK_DIM];
	float sum=0.0f;
	for(unsigned int tile = 0;tile < COARSE_FACTOR*2; tile++)
	{
		sum += input[i + tile*BLOCK_DIM];
	}

	input_s[threadIdx.x]=sum;
	__syncthreads();

	for(unsigned int stride = BLOCK_DIM/2; stride >0; stride /=2){
		if(threadIdx.x < stride)
		{
			input_s[threadIdx.x] +=input_s[threadIdx.x+stride];
		}
		__syncthreads();
	}
	if(threadIdx.x==0){
		partialSums[blockIdx.x] = input_s[threadIdx.x];
	}
}




float reduce_gpu(float* input, unsigned int N){
	

	//Allocate memory 
	
	float* input_d;	
	float* partialSums_d;
	float sum;

	hipMalloc((void**) &input_d, N*sizeof(float));
	hipDeviceSynchronize();
	
	//Copy data to GPU 
	hipMemcpy(input_d, input, N*sizeof(float), hipMemcpyHostToDevice);
	hipDeviceSynchronize();
	
	//Allocate partial sums
	const unsigned int numThreadsPerBlock = BLOCK_DIM;
	//const unsigned int numElementsPerBlock = 2*numThreadsPerBlock*COARSE_FACTOR;
	const unsigned int numElementsPerBlock = numThreadsPerBlock/4 * COARSE_FACTOR;
	const unsigned int numBlocks = (N + numElementsPerBlock -1)/numElementsPerBlock;
	float* partialSums = (float*) malloc(numBlocks*sizeof(float));

	hipMalloc((void**) &partialSums_d, numBlocks*sizeof(float));
	hipDeviceSynchronize();
	

	//Call kernel 
//	reduce_kernel<<< numBlocks, numThreadsPerBlock >>>(input_d, partialSums_d, N);
//	hipDeviceSynchronize();
	
	//reduce_kernel_controldiv<<< numBlocks, numThreadsPerBlock >>>(input_d, partialSums_d, N);
	//hipDeviceSynchronize();

	//reduce_kernel_sharedmem<<< numBlocks, numThreadsPerBlock >>>(input_d, partialSums_d, N);
	//hipDeviceSynchronize();

	reduce_kernel_sharedmem_coarsefactor<<< numBlocks, numThreadsPerBlock >>>(input_d, partialSums_d, N);
	hipDeviceSynchronize();

	//Copy data from GPU
	hipMemcpy(partialSums, partialSums_d, numBlocks*sizeof(float), hipMemcpyDeviceToHost);
	hipDeviceSynchronize();
	

	//Reduce partial Sums on the CPU
	for(unsigned int i=0; i<numBlocks; i++){
		sum+=partialSums[i];
	}

	//Free memory
	hipFree(input_d);
	free(partialSums);
	hipFree(partialSums_d);
	hipDeviceSynchronize();
	

	return sum;
}

int main(){
	int arraySize=65536;
	float a[arraySize];
	srand(time(0));
	float totalSum;

	for(int i = 0; i < arraySize; ++i)
	{
		a[i] = 1.0f + (float)(rand() % 100);
	}

	totalSum = reduce_gpu(a,arraySize);

	printf("Total Sum of the array: %f", totalSum);

}